#include "hip/hip_runtime.h"

/*
 * Skeleton function for Parallel Computing Class, 
 * Assignment: K-Means Algorithm (CUDA)
 *
 * To students: You should finish the implementation of k_means algorithm.
 *              You should add device functions/kernels to perform k_means on 
 *              GPU. The "k_means" function in this file is just an interface
 *              for passing in basic parameters needed.. You need to add GPU 
 *              kernels and launch them in the "k_means" function.
 *
 *              Note that the "k_means" function has two input parameters for
 *              block count and thread count per block. Please use these two
 *              parameters when launching your kernels.
 * 
 * Author:
 *     Wei Wang <wei.wang@utsa.edu>
 */
#include <stdio.h>
#include <float.h>
#include <math.h>
#include "k_means.h"

__global__ void assign_points_to_clusters(struct point *p, struct point *u, int *c, int k, int m) {
	__shared__ int atomic_c_point;
	int c_point;
	int c_cluster;
	if (threadIdx.x == 0)
		atomic_c_point = -1;
	__syncthreads();

	/* find the nearest center to each point */
	for (c_point = atomicAdd(&atomic_c_point, 1); c_point < m; c_point = atomicAdd(&atomic_c_point, 1))
	{
		double min_dist = DBL_MAX;
		struct point p1 = p[c_point];

		/* find the cluster that the point belongs to */
		for (c_cluster = 0; c_cluster < k; c_cluster++)
		{
			double dx = p1.x - u[c_cluster].x;
			double dy = p1.y - u[c_cluster].y;
			double dist = dx * dx + dy * dy;
			if (dist < min_dist)
			{
				/* Set the new minimum distance and assign the point to the current cluster */
				min_dist = dist;
				c[c_point] = c_cluster;
			}
		}
	}
	return;
}

__global__ void update_centers(struct point *p, struct point *u, int* c, int k, int m) {
	__shared__ int atomic_c_cluster;
	__shared__ int counter;
	int c_cluster;
	int c_point;
	if (threadIdx.x == 0) {
		atomic_c_cluster = -1;
	}
	__syncthreads();

	/* update the center for each cluster */
	for (c_cluster = atomicAdd(&atomic_c_cluster, 1); c_cluster < k; c_cluster = atomicAdd(&atomic_c_cluster, 1))
	{
		double sumx = 0;
		double sumy = 0;
		int cluster_size = 0;

		for (c_point = 0; c_point < m; c_point++)
		{
			if (c[c_point] == c_cluster)
			{
				sumx += p[c_point].x;
				sumy += p[c_point].y;
				cluster_size++;
			}
		}

		if (cluster_size > 0)
		{
			u[c_cluster].x = sumx / cluster_size;
			u[c_cluster].y = sumy / cluster_size;
		}
		else
		{
			int idx = atomicAdd(&counter, 1)%m;
			u[c_cluster] = p[idx];
		}
	}
	return;
}

/*
 * k_means: k_means clustering algorithm implementation.
 *
 * Input parameters:
 *     struct point p[]: array of data points
 *     int m           : number of data points in p[]
 *     int k           : number of clusters to find
 *     int iters       : number of clustering iterations to run
 *     int block_cnt   : number of blocks to use
 *     int threads_per_block: number of threads per block
 *
 * Output parameters:   
 *     struct point u[]: array of cluster centers
 *     int c[]         : cluster id for each data points
 */
void k_means(struct point p[MAX_POINTS], 
	     int m, 
	     int k,
	     int iters,
	     struct point u[MAX_CENTERS],
	     int c[MAX_POINTS],
	     int block_cnt,
	     int threads_per_block)
{
	int j; /* counter for random generator loop */
	int c_iter;

	struct point 	*d_p;		/* device points; size(m)*/
	struct point 	*d_u; 		/* device cluster centers; size(k) */
	int 			*d_c;		/* cluster id for each point; size(m) */
	size_t size_p = sizeof(struct point) * m;
	size_t size_u = sizeof(struct point) * k;
	size_t size_c = sizeof(int) * m;
	
	/* randomly initialized the centers */
	for(j = 0; j < k; j++)
		u[j] = random_center(p);

	/* Allocate space for device copies */
	hipMalloc((void **)&d_p, size_p);
	hipMalloc((void **)&d_u, size_u);
	hipMalloc((void **)&d_c, size_c);

	/* Copy input from host to device */
	hipMemcpy(d_p, p, size_p, hipMemcpyHostToDevice);
	hipMemcpy(d_u, u, size_u, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size_c, hipMemcpyHostToDevice);

	for (c_iter = 0; c_iter < iters; c_iter++) {
		/* block_cnt and threads_per_block */
		assign_points_to_clusters<<<block_cnt, threads_per_block>>>(d_p, d_u, d_c, k, m);
		update_centers<<<block_cnt, threads_per_block>>>(d_p, d_u, d_c, k, m);
	}

	/* Copy results from device to host */
	hipMemcpy(u, d_u, size_u, hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

	/* Cleanup */
	hipFree(d_p);
	hipFree(d_u);
	hipFree(d_c);

  	return;
}
