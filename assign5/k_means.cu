#include "hip/hip_runtime.h"

/*
 * Skeleton function for Parallel Computing Class, 
 * Assignment: K-Means Algorithm (CUDA)
 *
 * To students: You should finish the implementation of k_means algorithm.
 *              You should add device functions/kernels to perform k_means on 
 *              GPU. The "k_means" function in this file is just an interface
 *              for passing in basic parameters needed.. You need to add GPU 
 *              kernels and launch them in the "k_means" function.
 *
 *              Note that the "k_means" function has two input parameters for
 *              block count and thread count per block. Please use these two
 *              parameters when launching your kernels.
 * 
 * Author:
 *     Wei Wang <wei.wang@utsa.edu>
 */
#include <stdio.h>
#include <float.h>
#include <math.h>
#include "k_means.h"

struct params {
	int work_m;
	int work_k;
	int k;
	int m;
};

__device__ static int counter;

__device__ struct point random_center(struct point *p, int m) {
	int idx = (counter++) % m;
	return p[idx];
}

__global__ void assign_points_to_clusters(struct point *p, struct point *u, int *c, struct params* params) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int k = params->k;
	int m = params->m;
	int p_start = index * params->work_m;
	int p_end = p_start + params->work_m;
	if (p_end >= m)
		p_end = m;

	int c_point;
	int c_cluster;

	/* find the nearest center to each point */
	for (c_point = p_start; c_point < p_end; c_point++)
	{
		double min_dist = DBL_MAX;
		struct point p1 = p[c_point];

		/* find the cluster that the point belongs to */
		for (c_cluster = 0; c_cluster < k; c_cluster++)
		{
			double dx = p1.x - u[c_cluster].x;
			double dy = p1.y - u[c_cluster].y;
			double dist = dx * dx + dy * dy;
			if (dist < min_dist)
			{
				/* Set the new minimum distance and assign the point to the current cluster */
				min_dist = dist;
				c[c_point] = c_cluster;
			}
		}
	}
	return;
}

__global__ void update_centers(struct point *p, struct point *u, int* c, struct params *params) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int k = params->k;
	int m = params->m;
	int k_start = index * params->work_k;
	int k_end = k_start + params->work_k;
	if (k_end >= k)
		k_end = k;

	int c_cluster;
	int c_point;

	/* update the center for each cluster */
	for (c_cluster = k_start; c_cluster < k_end; c_cluster++)
	{
		double sumx = 0;
		double sumy = 0;
		int cluster_size = 0;

		for (c_point = 0; c_point < m; c_point++)
		{
			if (c[c_point] == c_cluster)
			{
				sumx += p[c_point].x;
				sumy += p[c_point].y;
				cluster_size += 1;
			}
		}

		if (cluster_size > 0)
		{
			u[c_cluster].x = sumx / cluster_size;
			u[c_cluster].y = sumy / cluster_size;
		}
		else
		{
			u[c_cluster] = random_center(p, m);
		}
	}
	return;
}

/*
 * k_means: k_means clustering algorithm implementation.
 *
 * Input parameters:
 *     struct point p[]: array of data points
 *     int m           : number of data points in p[]
 *     int k           : number of clusters to find
 *     int iters       : number of clustering iterations to run
 *     int block_cnt   : number of blocks to use
 *     int threads_per_block: number of threads per block
 *
 * Output parameters:   
 *     struct point u[]: array of cluster centers
 *     int c[]         : cluster id for each data points
 */
void k_means(struct point p[MAX_POINTS],
	     int m,
	     int k,
	     int iters,
	     struct point u[MAX_CENTERS],
	     int c[MAX_POINTS],
	     int block_cnt,
	     int threads_per_block)
{
	int j; /* counter for random generator loop */
	int c_iter;

	struct point 	*d_p;		/* device points; size(m)*/
	struct point 	*d_u; 		/* device cluster centers; size(k) */
	int 			*d_c;		/* cluster id for each point; size(m) */
	struct params 	*d_params;	/* parameters to tell how much work to do */
	size_t size_p = sizeof(struct point) * m;
	size_t size_u = sizeof(struct point) * k;
	size_t size_c = sizeof(int) * m;
	size_t size_params = sizeof(struct params);

	/* randomly initialized the centers */
	for(j = 0; j < k; j++)
		u[j] = random_center(p);

	/* Allocate space for device copies */
	hipMalloc((void **)&d_p, size_p);
	hipMalloc((void **)&d_u, size_u);
	hipMalloc((void **)&d_c, size_c);
	hipMalloc((void **)&d_params, size_params);

	struct params params;
	params.work_m = ceil((float)m / (block_cnt * threads_per_block));
	params.work_k = ceil((float)k / (block_cnt * threads_per_block));
	params.k = k;
	params.m = m;

	/* Copy input from host to device */
	hipMemcpy(d_p, p, size_p, hipMemcpyHostToDevice);
	hipMemcpy(d_u, u, size_u, hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, size_c, hipMemcpyHostToDevice);
	hipMemcpy(d_params, &params, size_params, hipMemcpyHostToDevice);

	for (c_iter = 0; c_iter < iters; c_iter++) {
		/* block_cnt and threads_per_block */
		assign_points_to_clusters<<<block_cnt, threads_per_block>>>(d_p, d_u, d_c, d_params);
		update_centers<<<block_cnt, threads_per_block>>>(d_p, d_u, d_c, d_params);
	}

	/* Copy results from device to host */
	hipMemcpy(u, d_u, size_u, hipMemcpyDeviceToHost);
	hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

	/* Cleanup */
	hipFree(d_p);
	hipFree(d_u);
	hipFree(d_c);
	hipFree(d_params);

  	return;
}

